#include "hip/hip_runtime.h"
//
// Created by tony on 12/06/24.
//
#include <torch/torch.h>
#include <ATen/ATen.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// Utility function to check CUDA errors
inline void checkCudaError(hipError_t result, char const *const func, const char *const file, int const line) {
  if (result != hipSuccess) {
    std::cerr << "CUDA error = " << static_cast<int>(result) << " at " <<
              file << ":" << line << " '" << func << "' \n" << hipGetErrorString(result) << std::endl;
    exit(1);
  }
}

#define CHECK_CUDA_ERROR(val) checkCudaError((val), #val, __FILE__, __LINE__)
__global__ void matrixMulCUDA(float *a, float *b, float *c, int M, int N, int K) {
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  if (row < M && col < K) {
    float sum = 0.0;
    for (int i = 0; i < N; i++) {
      sum += a[row * N + i] * b[i * K + col];
    }
    c[row * K + col] = sum;
  }
}

torch::Tensor CudaMM(torch::Tensor a, torch::Tensor b) {
  // Ensure input tensors are on the GPU and are contiguous
  a = a.to(at::kCUDA).contiguous();
  b = b.to(at::kCUDA).contiguous();

  // Dimensions
  int M = a.size(0);
  int N = a.size(1);
  int K = b.size(1);

  // Create the output tensor on the GPU

  torch::Tensor c = torch::zeros({M, K}).to(at::kCUDA);

  // Define block and grid sizes
  dim3 threadsPerBlock(4, 4);
  dim3 blocksPerGrid((K + threadsPerBlock.x - 1) / threadsPerBlock.x,
                     (M + threadsPerBlock.y - 1) / threadsPerBlock.y);

  // Launch the kernel
  matrixMulCUDA<<<blocksPerGrid, threadsPerBlock>>>(a.data_ptr<float>(),
                                                    b.data_ptr<float>(),
                                                    c.data_ptr<float>(),
                                                    M,
                                                    N,
                                                    K);

  // Wait for GPU to finish before accessing on host
  CHECK_CUDA_ERROR(hipDeviceSynchronize());

  // Transfer the result tensor to the CPU
  return c.to(torch::kCPU);
}